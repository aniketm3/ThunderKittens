#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>
#include <tuple>

#ifdef TORCH_COMPILE
#define TK_COMPILE_LIN_ATTN
#endif

//RTX4090
//D=16 => NUM_WORKERS 16 ACTIVE_TILES 8 is ok
//D=64 => NUM_WORKERS 8 ACTIVE_TILES 4 is ok
//D=128 => NUM_WORKERS 2 ACTIVE_TILES 1 is ok

#define NUM_WORKERS 8 //16
#define ACTIVE_TILES 4 //8
#define NUM_THREADS NUM_WORKERS*kittens::WARP_THREADS

#define ROWS 16
#define ATTN_D 64
#define BETA 0.5f //defing the beta weighting the delta update

using namespace kittens;

// do a cumsum on a tile, starting from some given position total_block_idx
// with [2, 1, 8, 3] and total_block_idx = 2, it will give [13, 14, 8, 11] (loops back)
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void cumsum_inplace(ST (&x)[N_TILES], int total_block_idx) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;

    for(int i = 1; i < N_TILES; i++) {
        #pragma unroll
        for(int j = threadIdx.x; j < ST::num_elements; j+=STRIDE) {
            x[(total_block_idx+i)%N_TILES].data[j] += x[(total_block_idx+i-1)%N_TILES].data[j];
        }
    }
}

template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void revcumsum_inplace(ST (&x)[N_TILES], int total_block_idx) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;

    for(int i = N_TILES-1; i > 0; i--) {
        #pragma unroll
        for(int j = threadIdx.x; j < ST::num_elements; j+=STRIDE) {
            x[(total_block_idx+i)%N_TILES].data[j] += x[(total_block_idx+i+1)%N_TILES].data[j];
        }
    }
}

// ---------------------------------------------------------------------------------------------------
// ----------------------------------------- Forward kernel ------------------------------------------
// ---------------------------------------------------------------------------------------------------

struct fwd_globals {
    using q_tile = st_bf<ROWS, ATTN_D>;
    using k_tile = st_bf<ROWS, ATTN_D>;
    using v_tile = st_bf<ROWS, ATTN_D>;
    using o_tile = st_bf<ROWS, ATTN_D>;

    // global layouts
    using q_gl     = gl<bf16,  -1, -1, -1, ATTN_D, q_tile>;
    using k_gl     = gl<bf16,  -1, -1, -1, ATTN_D, k_tile>;
    using v_gl     = gl<bf16,  -1, -1, -1, ATTN_D, v_tile>;
    using o_gl     = gl<bf16,  -1, -1, -1, ATTN_D, o_tile>;

    // pointers
    q_gl q;
    k_gl k;
    v_gl v;
    o_gl o;

    long unsigned int n;
};

__global__ __launch_bounds__(NUM_THREADS, 1)
void delta_attention_fwd(const __grid_constant__ fwd_globals g) {

    const int batch = blockIdx.y;
    const int head  = blockIdx.x;

    int warpid = kittens::warpid(); 

    extern __shared__ alignment_dummy __shm[]; 
    shared_allocator al((int*)&__shm[0]);

    st_bf<ROWS, ATTN_D> (&qo_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    st_bf<ROWS, ATTN_D> (&k_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    st_bf<ROWS, ATTN_D> (&v_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    st_bf<ATTN_D, ATTN_D> (&s_s)[ACTIVE_TILES + 1]  = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES + 1>();

    int total_block_idx = 0;

    if (warpid < ACTIVE_TILES + 1) {
        zero(s_s[warpid]);
    }

    int n_blocks = g.n / (ACTIVE_TILES * ROWS); // number of chunks we will loop over

    for (int block = 0; block < n_blocks; block++) {
        rt_bf<ROWS, ATTN_D> q, k;
        // rt_bf<ATTN_D, ROWS> kt;
        // rt_bf<ROWS, ROWS> local_attn_bf;
        // rt_fl<ROWS, ROWS> local_attn;
        rt_bf<ROWS, ATTN_D> v;
        // rt_fl<ATTN_D, ATTN_D> accum;
        rt_fl<ROWS, ATTN_D> o;

        rt_fl<ATTN_D, ATTN_D> s_state; // current memory state loaded in
        rt_fl<ATTN_D, ATTN_D> s_new; // new memory state to place in
        rt_fl<ROWS, ATTN_D> error;
        rt_fl<ROWS, ATTN_D> beta_error;
        rt_fl<ROWS, ATTN_D> P;
        rt_fl<ATTN_D, ATTN_D> delta;


        int cur_idx;

        if (warpid < ACTIVE_TILES) {
            cur_idx = block * ACTIVE_TILES + warpid;
            load(qo_s[warpid], g.q, {batch, head, cur_idx, 0});
            load(k_s[warpid], g.k, {batch, head, cur_idx, 0});
        }
        else {
            cur_idx = block * ACTIVE_TILES + warpid - ACTIVE_TILES;
            load(v_s[warpid - ACTIVE_TILES], g.v, {batch, head, cur_idx, 0});
        }
        
        __syncthreads();

        //implementing s_t = s_t-1 - beta(s_t-1 * k_t - v_t) O* k_t
        if (warpid < ACTIVE_TILES) {
            load(q, qo_s[warpid]);
            load(k, k_s[warpid]);
            
            zero(s_state);
            load(s_state, s_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]); //loading current memory state

            matvec_tile(P, s_state, k); // compute P <- s_state * k(i,:)^T for each of the rows i

            load(v, v_s[warpid]);

            copy(error, P); // error <- s_state * k(i,:)^T
            sub(error, error, v); // error <- s_state * k(i,:)^T - v(i,:)

            copy(beta_error, error);
            mul(beta_error, beta_error, BETA); // beta_error <- beta * (s_state * k(i,:)^T - v(i,:))

            zero(delta); 
            // computing the delta  value doing the outer product of the error and the k value
            for (int i = 0; i < ROWS; i++) {
                rt_fl<1, ATTN_D> k_row, error_row;

                get_row(k_row, k, i);
                get_row(error_row, beta_error, i);
                rt_fl<ATTN_D, ATTN_D> outer;
                outer_product(outer, error_row, k_row);
                add(delta, delta, outer);
            }

            copy(s_new, s_state);
            sub(s_new, s_new, delta); // s_new <- s_state - delta

            store(s_s[(total_block_idx + warpid + 1) % (ACTIVE_TILES + 1)], s_new); // storing the new memory state

            zero(o);
            matvec_tile(o, s_new, q); // compute o <- s_new * q(i,:)^T for each of the rows i

            //do i store this thread's output to the qo_s[warpid]?
            store(qo_s[warpid], o);
        }

        __syncthreads();
        cumsum_inplace<NUM_WORKERS>(s_s, total_block_idx);
        __syncthreads();

        if (warpid < ACTIVE_TILES) {
            rt_bf<ROWS, ATTN_D> s;
            load(q, qo_s[warpid]);
            load(s, s_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
            auto &s_col = kittens::swap_layout_inplace(s);

            // do i need to laod o again here? o is the same as q
            mma_AB(o, q, s_col, o);
            store(qo_s[warpid], o);
        }

        total_block_idx = (total_block_idx + ACTIVE_TILES) % (ACTIVE_TILES + 1);
        __syncthreads();

        if(warpid < ACTIVE_TILES) {
            store(g.o, qo_s[warpid], {batch, head, cur_idx, 0});
        }
        __syncthreads();
    }
}

fwd_globals fwd_init(
    bf16 *d_q, bf16 *d_k, bf16 *d_v,
    bf16 *d_o,
    long unsigned int ATTN_B, long unsigned int ATTN_H, long unsigned int ATTN_N
) {
    // global pointers

    using globals = fwd_globals;

    using q_tile     = globals::q_tile;
    using k_tile     = globals::k_tile;
    using v_tile     = globals::v_tile;
    using o_tile     = globals::o_tile;

    // global layouts
    using q_gl     = globals::q_gl;
    using k_gl     = globals::k_gl;
    using v_gl     = globals::v_gl;
    using o_gl     = globals::o_gl;

    q_gl     q_arg{d_q, ATTN_B, ATTN_H, ATTN_N, nullptr};
    k_gl     k_arg{d_k, ATTN_B, ATTN_H, ATTN_N, nullptr};
    v_gl     v_arg{d_v, ATTN_B, ATTN_H, ATTN_N, nullptr};
    o_gl     o_arg{d_o, ATTN_B, ATTN_H, ATTN_N, nullptr};

    globals g{
        q_arg, k_arg, v_arg, o_arg, ATTN_N
    };
    return g;
}

// ---------------------------------------------------------------------------------------------------
// ----------------------------------------- Backward kernel -----------------------------------------
// ---------------------------------------------------------------------------------------------------


struct bwd_globals {
    using q_tile = st_bf<ROWS, ATTN_D>;
    using k_tile = st_bf<ROWS, ATTN_D>;
    using v_tile = st_bf<ROWS, ATTN_D>;
    using do_tile = st_bf<ROWS, ATTN_D>;

    // global layouts
    using q_gl     = gl<bf16, -1, -1, -1, ATTN_D, q_tile>;
    using k_gl     = gl<bf16, -1, -1, -1, ATTN_D, k_tile>;
    using v_gl     = gl<bf16, -1, -1, -1, ATTN_D, v_tile>;
    using do_gl    = gl<bf16, -1, -1, -1, ATTN_D, do_tile>;

    // pointers
    q_gl q;
    k_gl k;
    v_gl v;
    do_gl d_o;

    q_gl dq;
    k_gl dk;
    v_gl dv;

    long unsigned int n;
};

__global__ __launch_bounds__(NUM_THREADS, 1)
void delta_attention_bwd(const __grid_constant__ bwd_globals g) {
    
    const int batch = blockIdx.y;
    const int head  = blockIdx.x;

    int warpid = kittens::warpid(); 

    extern __shared__ alignment_dummy __shm[]; 
    shared_allocator al((int*)&__shm[0]);

    st_bf<ROWS, ATTN_D> (&dodqqdk_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // do,dq for 1st loop, q,dk for 2nd loop
    st_bf<ROWS, ATTN_D> (&k_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // k for 1st and 2nd
    st_bf<ROWS, ATTN_D> (&v_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // v for 1st and 2nd
    st_bf<ATTN_D, ATTN_D> (&hidden_s)[ACTIVE_TILES + 1]  = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES + 1>(); // accumulates hidden states (memory state S_t form forward pass
    st_bf<ATTN_D, ATTN_D> (&dhidden_s)[ACTIVE_TILES + 1]  = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES + 1>(); // hidden state gradients (ds)
    st_bf<ROWS, ATTN_D> (&dodv_s)[ACTIVE_TILES] = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // do,dv for 2nd

    int total_block_idx = 0;

    if (warpid < ACTIVE_TILES + 1) {
        zero(hidden_s[warpid]); 
        zero(dhidden_s[warpid]);
    }
    
    int n_blocks = g.n / (ACTIVE_TILES * ROWS);

    // first loop: dq
    for (int block = n_blocks - 1; block >= 0; block--) { // iterate backwards since S_t+1 impacts S_t
        rt_bf<ROWS, ATTN_D> d_o, k, v;
        rt_bf<ATTN_D, ROWS> vt;
        rt_bf<ROWS, ROWS> local_attn_bf; //less precision long term storage
        rt_fl<ROWS, ROWS> local_attn;
        rt_fl<ATTN_D, ATTN_D> d_accum;
        rt_fl<ROWS, ATTN_D> dq;

        int cur_idx;

        // load the data -> first half k, do, second half v 
        if(warpid < ACTIVE_TILES) {
            cur_idx = block * ACTIVE_TILES + warpid;
            load(dodqqdk_s[warpid], g.d_o, {batch, head, cur_idx, 0});
            load(k_s[warpid], g.k, {batch, head, cur_idx, 0});
        }
        else {
            cur_idx = block * ACTIVE_TILES + warpid - ACTIVE_TILES;
            load(v_s[warpid - ACTIVE_TILES], g.v, {batch, head, cur_idx, 0});
        }
        __syncthreads();

        if (warpid < ACTIVE_TILES) {
            load(d_o, dodqqdk_s[warpid]); // using dodqqdk_s as d_o
            load(v, v_s[warpid]);

            zero(local_attn);
            mma_ABt(local_attn, d_o, v, local_attn); // local_attn <- d_o * v^T

            // calculating the decay factor as a mask
            rt_bf<ROWS, ROWS> decay;
            zero(decay);

            //calculaing the decay exponent as a lower triangular matrix
            bf16* decay_ptr = reinterpret_cast<bf16*>(&decay);
            #pragma unroll
            for (int i = 0; i < ROWS; i++) {
                for (int j=0; j<ROWS; j++) {
                    // decay.data[i*ROWS + j] *= __float2bfloat16(powf(BETA, i-j));
                    decay_ptr[i * ROWS + j] = __float2bfloat16(powf(BETA, (float)(i - j)));
                }
            }

            // applying the decay factor to the local attention
            bf16* local_attn_ptr = reinterpret_cast<bf16*>(&local_attn);
            #pragma unroll
            for (int i = 0; i < ROWS; i++) {
                for (int j=0; j <ROWS; j++) {
                    local_attn_ptr[i * ROWS + j] *= decay_ptr[i * ROWS + j];
                }
            }
            
            //calculate local intra attention (will later accumulate over everything else)
            zero(dq);
            copy(local_attn_bf, local_attn);
            load(k, k_s[warpid]);
            auto &k_col = swap_layout_inplace(k);

            mma_AB(dq, local_attn_bf, k_col, dq); // dq <- local_attn * k^T
            
            //find dS
            zero(d_accum);
            mma_AtB(d_accum, v, k_col, d_accum); //or seperatly transpose v

            //add gradient to the accumulated gradient
            rt_fl<ATTN_D, ATTN_D> d_accum_loaded;
            load(d_accum_loaded, d_accum[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);

            bf16* d_accum_ptr = reinterpret_cast<bf16*>(&d_accum);
            bf16* d_accum_loaded_ptr = reinterpret_cast<bf16*>(&d_accum_loaded);
            #pragma unroll
            for (int i=0; i<ATTN_D; i++) {
                for (int j=0; j<ATTN_D; j++) {
                    d_accum_ptr[i*ATTN_D + j] += d_accum_ptr[i*ATTN_D + j] * BETA + d_accum_loaded_ptr[i*ATTN_D + j];
                }
            }

            store(dhidden_s[(total_block_idx + warpid + 1) % (ACTIVE_TILES + 1)], d_accum);
        }

        __syncthreads();
        revcumsum_inplace<NUM_WORKERS>(dhidden_s, total_block_idx);
        __syncthreads();

        // doing actual dq calculation over the accumulated gradients
        if (warpid < ACTIVE_TILES) {
            rt_bf<ATTN_D, ATTN_D> gradient;
            load(d_o, dodqqdk_s[warpid]);
            load(gradient, dhidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
            auto &gradient_col = swap_layout_inplace(gradient);
            mma_AB(dq, d_o, gradient_col, dq);
            store(dodqqdk_s[warpid], dq);
        }

        total_block_idx = (total_block_idx + ACTIVE_TILES) % (ACTIVE_TILES + 1);
        __syncthreads();
        
        if (warpid < ACTIVE_TILES) {
            store(g.dq, dodqqdk_s[warpid], {batch, head, cur_idx, 0});
        }
        __syncthreads();
    }

    // second loop: dk, dv
    total_block_idx = 0;

    if (warpid < ACTIVE_TILES + 1) {
        zero(hidden_s[warpid]);
    }

    for (int block = 0; block < n_blocks; block++) {
        rt_bf<ROWS, ATTN_D> d_o, q, k, v;
        rt_bf<ROWS, ATTN_D, col_l> q_col;
        rt_bf<ATTN_D, ROWS> qt;
        rt_bf<ROWS, ROWS> local_attn_bf;
        rt_fl<ROWS, ROWS> local_attn;
        rt_fl<ATTN_D, ATTN_D> dhidden_accum; // dS accumulated values
        rt_fl<ROWS, ATTN_D> dk, dv;

        int cur_idx;
        if (warpid < ACTIVE_TILES) {
            cur_idx = block * ACTIVE_TILES + warpid;
            load(dodqqdk_s[warpid], g.d_o, {batch, head, cur_idx, 0});
            load(k_s[warpid], g.k, {batch, head, cur_idx, 0});
        }
        else {
            cur_idx = block * ACTIVE_TILES + warpid - ACTIVE_TILES;
            load(v_s[warpid - ACTIVE_TILES], g.v, {batch, head, cur_idx, 0});
            load(dodv_s[warpid - ACTIVE_TILES], g.d_o, {batch, head, cur_idx, 0});
        }
        __syncthreads();

        if (warpid < ACTIVE_TILES) {
            load(d_o, dodv_s[warpid]);
            load(v, v_s[warpid]);

            // dk calculaiton start
            zero(local_attn);
            mma_ABt(local_attn, v, d_o, local_attn); // local_attn <- v * d_o^T

            rt_bf<ROWS, ROWS> decay;
            zero(decay);

            #pragma unroll
            for (int i = 0; i < ROWS; i++) {
                for (int j=0; j<=i; j++) {
                    decay.data[i*ROWS + j] *= powf(BETA, i-j)
                }
            }

            //applying mask to local delta calculated ealirer
            #pragma unroll
            for (int i = 0; i < ROWS; i++) {
                for (int j=0; j <ROWS; j++) {
                    local_attn.data[i * ROWS + j] *= decay.data[i * ROWS + j];
                }
            }

            copy(local_attn_bf, local_attn);
            load(q, dodqqdk_s[warpid]);
            swap_layout(q_col, q);

            zero(dk);
            mma_AB(dk, local_attn_bf, q_col, dk); // dk <- local_attn * q^T 

            // dv calculation start
            load(k, k_s[warpid]);
            zero(local_attn);
            mma_ABt(local_attn, k, q, local_attn); // local_attn <- k * q^T

            //applying decay
            #pragma unroll
            for (int i = 0; i < ROWS; i++) {
                for (int j=0; j< ROWS; j++) {
                    local_attn.data[i * ROWS + j] *= decay.data[i * ROWS + j];
                }
            }

            copy(local_attn_bf, local_attn);

            zero(dv);
            auto &d_o_col = swap_layout_inplace(d_o);
            mma_AB(dv, local_attn_bf, d_o_col, dv); // dv <- local_attn * d_o^T

            // calculating S_t (hidden state)
            transpose_sep(qt, q);
            zero(dhidden_accum);
            mma_AB(dhidden_accum, qt, d_o_col, dhidden_accum); // hidden_s <- q^T * d_o

            //again applying decay factor
            rt_fl<ATTN_D, ATTN_D> hidden_loaded;
            load(hidden_loaded, dhidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);

            #pragma unroll
            for (int i=0; i<ATTN_D; i++) {
                for (int j=0; j<ATTN_D; j++) {
                    dhidden_accum.data[i*ATTN_D + j] += hidden_loaded.data[i*ATTN_D + j] * BETA + dhidden_accum.data[i*ATTN_D + j];
                }
            }
            store(hidden_s[(total_block_idx + warpid + 1) % (ACTIVE_TILES + 1)], dhidden_accum);
        }

        __syncthreads();
        cumsum_inplace<NUM_WORKERS>(hidden_s, total_block_idx);
        __syncthreads();

        // next part of dk calculation
        if (warpid < ACTIVE_TILES) {
            rt_bf<ATTN_D, ATTN_D> gradient;
            load(v, v_s[warpid]);
            load(gradient, hidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
            auto &gradient_col = swap_layout_inplace(gradient);
            mma_ABt(dk, v, gradient_col, dk);
            store(dodqqdk_s[warpid], dk);
        }

        __syncthreads();

        // next part of dv calculation
        if (warpid < ACTIVE_TILES) {
            rt_bf<ATTN_D, ATTN_D> gradient;
            load(k, k_s[warpid]);
            load(gradient, hidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
            auto &gradient_col = swap_layout_inplace(gradient);
            mma_AB(dv, k, gradient, dv);
            store(dodv_s[warpid], dv);
        }

        total_block_idx = (total_block_idx + ACTIVE_TILES) % (ACTIVE_TILES + 1);
        __syncthreads();

        if (warpid < ACTIVE_TILES) {
            store(g.dk, dodqqdk_s[warpid], {batch, head, cur_idx, 0});
            store(g.dv, dodv_s[warpid], {batch, head, cur_idx, 0});
        }
        __syncthreads();
    }

}

bwd_globals bwd_init(
    bf16 *d_q, bf16 *d_k, bf16 *d_v, bf16 *d_do,
    bf16 *d_dq, bf16 *d_dk, bf16 *d_dv,
    long unsigned int ATTN_B, long unsigned int ATTN_H, long unsigned int ATTN_N
) {
    // global pointers

    using globals = bwd_globals;

    using q_tile     = globals::q_tile;
    using k_tile     = globals::k_tile;
    using v_tile     = globals::v_tile;
    using do_tile     = globals::do_tile;

    // global layouts
    globals::q_gl  q_arg{d_q, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::k_gl  k_arg{d_k, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::v_gl  v_arg{d_v, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::do_gl do_arg{d_do, ATTN_B, ATTN_H, ATTN_N, nullptr};

    globals::q_gl dq_arg{d_dq, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::k_gl dk_arg{d_dk, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::v_gl dv_arg{d_dv, ATTN_B, ATTN_H, ATTN_N, nullptr};

    globals g{
        q_arg, k_arg, v_arg, do_arg, dq_arg, dk_arg, dv_arg, ATTN_N
    };
    return g;
}


#ifdef TK_COMPILE_LIN_ATTN
#include "pyutils/torch_helpers.cuh"
#include <iostream>
void dispatch_fwd( 
    bf16 *d_q, bf16 *d_k, bf16 *d_v, bf16 *d_o,
    int ATTN_B, int ATTN_H, int ATTN_N
){
    fwd_globals g = fwd_init(
        d_q, d_k, d_v,
        d_o,
        ATTN_B, ATTN_H, ATTN_N
    );

    // launch
    unsigned long mem_size = 100000; // 4090
    hipDeviceSynchronize();
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        delta_attention_fwd),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    dim3 grid(ATTN_H, ATTN_B);
    delta_attention_fwd<<<grid,NUM_THREADS,mem_size>>>(g);
    CHECK_CUDA_ERROR(hipGetLastError());
    hipDeviceSynchronize();
}

torch::Tensor delta_attn_forward(
    const torch::Tensor q, 
    const torch::Tensor k,
    const torch::Tensor v
) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);

    int B = q.size(0);
    int H = q.size(1);
    int DV = v.size(3);
    int N  = q.size(2);
    int FD = k.size(3);

    // checks
    TORCH_CHECK(k.size(0) == B, "k batch?");
    TORCH_CHECK(k.size(1) == H, "k heads?");
    TORCH_CHECK(k.size(2) == N, "k length?");

    TORCH_CHECK(v.size(0) == B, "v batch?");
    TORCH_CHECK(v.size(1) == H, "v heads?");
    TORCH_CHECK(v.size(2) == N, "v length?");

    // allocate output
    torch::Tensor out = torch::empty({B, H, N, DV}, v.options());

    // convert to bf16
    c10::BFloat16 *q_bf16 = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_bf16 = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_bf16 = v.data_ptr<c10::BFloat16>();
    
    bf16 *d_q = reinterpret_cast<bf16*>(q_bf16);
    bf16 *d_k = reinterpret_cast<bf16*>(k_bf16);
    bf16 *d_v = reinterpret_cast<bf16*>(v_bf16);
    bf16 *d_o = reinterpret_cast<bf16*>(out.data_ptr<c10::BFloat16>());

    dispatch_fwd(
        d_q, d_k, d_v, d_o,
        B, H, N
    );

    CHECK_CUDA_ERROR(hipGetLastError());
    return out;
    hipDeviceSynchronize();
}

void dispatch_bwd(
    bf16 *d_q, bf16 *d_k, bf16 *d_v, bf16 *d_do,
    bf16 *d_dq, bf16 *d_dk, bf16 *d_dv,
    int ATTN_B, int ATTN_H, int ATTN_N
){
    bwd_globals g = bwd_init(
        d_q, d_k, d_v, d_do,
        d_dq, d_dk, d_dv,
        ATTN_B, ATTN_H, ATTN_N
    );

    // launch
    unsigned long mem_size = 100000; // 4090
    hipDeviceSynchronize();
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        delta_attention_bwd),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    dim3 grid(ATTN_H, ATTN_B);
    delta_attention_bwd<<grid,NUM_THREADS,mem_size>>>(g);
    CHECK_CUDA_ERROR(hipGetLastError());
    hipDeviceSynchronize();
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> delta_attn_backward(
    const torch::Tensor q, 
    const torch::Tensor k,
    const torch::Tensor v,
    const torch::Tensor _do
) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(_do);

    int B = q.size(0);
    int H = q.size(1);
    int DV = v.size(3);
    int N  = q.size(2);
    int FD = k.size(3);

    // checks
    TORCH_CHECK(k.size(0) == B, "k batch?");
    TORCH_CHECK(k.size(1) == H, "k heads?");
    TORCH_CHECK(k.size(2) == N, "k length?");

    TORCH_CHECK(v.size(0) == B, "v batch?");
    TORCH_CHECK(v.size(1) == H, "v heads?");
    TORCH_CHECK(v.size(2) == N, "v length?");

    // allocate output
    torch::Tensor out_dq = torch::empty({B, H, N, FD}, q.options());
    torch::Tensor out_dk = torch::empty({B, H, N, FD}, k.options());
    torch::Tensor out_dv = torch::empty({B, H, N, DV}, v.options());

    // convert to bf16
    c10::BFloat16 *q_bf16 = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_bf16 = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_bf16 = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *do_bf16 = _do.data_ptr<c10::BFloat16>();
    
    bf16 *d_q = reinterpret_cast<bf16*>(q_bf16);
    bf16 *d_k = reinterpret_cast<bf16*>(k_bf16);
    bf16 *d_v = reinterpret_cast<bf16*>(v_bf16);
    bf16 *d_do = reinterpret_cast<bf16*>(do_bf16);
    bf16 *d_dq = reinterpret_cast<bf16*>(out_dq.data_ptr<c10::BFloat16>());
    bf16 *d_dk = reinterpret_cast<bf16*>(out_dk.data_ptr<c10::BFloat16>());
    bf16 *d_dv = reinterpret_cast<bf16*>(out_dv.data_ptr<c10::BFloat16>());

    dispatch_bwd(
        d_q, d_k, d_v, d_do,
        d_dq, d_dk, d_dv,
        B, H, N
    );

    CHECK_CUDA_ERROR(hipGetLastError());
    return std::make_tuple(out_dq, out_dk, out_dv);
    hipDeviceSynchronize();
}

// #else
// #ifdef FWD_HARNESS
// #include "4090_harness_fwd.impl"
// #else
// #include "4090_harness_bwd.impl"
// #endif
#endif