#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>
#include <tuple>
#include <iostream>

#ifdef TORCH_COMPILE
#define TK_COMPILE_LIN_ATTN
#endif

//RTX4090
//D=16 => NUM_WORKERS 16 ACTIVE_TILES 8 is ok
//D=64 => NUM_WORKERS 8 ACTIVE_TILES 4 is ok
//D=128 => NUM_WORKERS 2 ACTIVE_TILES 1 is ok

#define NUM_WORKERS 8 //16 // TODO: do 8 warpid's
#define ACTIVE_TILES 8 //8
#define NUM_THREADS NUM_WORKERS*kittens::WARP_THREADS

#define ROWS 16

#undef ATTN_D
#define ATTN_D 16

#define ONE 16

#define BETA 0.01f //defing the beta weighting the delta update

using namespace kittens;

// do a cumsum on a tile, starting from some given position total_block_idx
// with [2, 1, 8, 3] and total_block_idx = 2, it will give [13, 14, 8, 11] (loops back)
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void cumsum_inplace(ST (&x)[N_TILES], int total_block_idx) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;

    for(int i = 1; i < N_TILES; i++) {
        #pragma unroll
        for(int j = threadIdx.x; j < ST::num_elements; j+=STRIDE) {
            x[(total_block_idx+i)%N_TILES].data[j] += x[(total_block_idx+i-1)%N_TILES].data[j];
        }
    }
}

template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void revcumsum_inplace(ST (&x)[N_TILES], int total_block_idx) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;

    for(int i = N_TILES-1; i > 0; i--) {
        #pragma unroll
        for(int j = threadIdx.x; j < ST::num_elements; j+=STRIDE) {
            x[(total_block_idx+i)%N_TILES].data[j] += x[(total_block_idx+i+1)%N_TILES].data[j];
        }
    }
}

// ---------------------------------------------------------------------------------------------------
// ----------------------------------------- Forward kernel ------------------------------------------
// ---------------------------------------------------------------------------------------------------

struct fwd_globals {
    using q_tile = st_bf<ATTN_D, ONE>;
    using k_tile = st_bf<ATTN_D, ONE>;
    using v_tile = st_bf<ATTN_D, ONE>;
    using o_tile = st_bf<ATTN_D, ONE>;
    // using q_tile = st_bf<1, ATTN_D>;
    // using k_tile = st_bf<1, ATTN_D>;
    // using v_tile = st_bf<1, ATTN_D>;
    // using o_tile = st_bf<1, ATTN_D>;

    // using error_tile = st_bf<ROWS, ATTN_D>;
    // using p_tile = st_bf<ROWS, ATTN_D>;
    // using s_state_tile = st_bf<ATTN_D, ATTN_D>;
    // using delta_state_tile = st_bf<ATTN_D, ATTN_D>;
    // using p_state_tile = st_bf<ATTN_D, ATTN_D>;


    // global layouts
    using q_gl     = gl<bf16,  -1, -1, -1, ATTN_D, q_tile>;
    using k_gl     = gl<bf16,  -1, -1, -1, ATTN_D, k_tile>;
    using v_gl     = gl<bf16,  -1, -1, -1, ATTN_D, v_tile>;
    using o_gl     = gl<bf16,  -1, -1, -1, ATTN_D, o_tile>;

    // using error_tile = st_bf<ROWS, ATTN_D>;
    // using p_tile = st_bf<ROWS, ATTN_D>;
    // using s_state_tile = st_bf<ATTN_D, ATTN_D>;
    // using delta_state_tile = st_bf<ATTN_D, ATTN_D>;
    // using p_state_tile = st_bf<ATTN_D, ATTN_D>;



    // pointers
    q_gl q;
    k_gl k;
    v_gl v;
    o_gl o;



    long unsigned int n;
};

__global__ __launch_bounds__(NUM_THREADS, 1)
void delta_attention_fwd(const __grid_constant__ fwd_globals g) {

    const int batch = blockIdx.y;
    const int head  = blockIdx.x;

    int warpid = kittens::warpid(); 

    extern __shared__ alignment_dummy __shm[]; 
    shared_allocator al((int*)&__shm[0]);

    // TODO
    st_bf<ROWS, ATTN_D> (&qo_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    st_bf<ROWS, ATTN_D> (&k_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    st_bf<ROWS, ATTN_D> (&v_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    st_bf<ATTN_D, ATTN_D> (&s_s)[ACTIVE_TILES + 1]  = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES + 1>();

    // st_bf<ROWS, ATTN_D> (&error_s)[ACTIVE_TILES] = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    // st_bf<ROWS, ATTN_D> (&p_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>();
    // st_bf<ATTN_D, ATTN_D> (&s_state_s)[ACTIVE_TILES]   = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES>();
    // st_bf<ATTN_D, ATTN_D> (&delta_s)[ACTIVE_TILES]   = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES>();
    // st_bf<ATTN_D, ATTN_D> (&s_new_s)[ACTIVE_TILES]   = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES>();

    // st_bf<ROWS, ATTN_D> (&shared_debug)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); //shared tile for debugging
    // st_bf<ATTN_D, ROWS> (&shared_debug_T)[ACTIVE_TILES]   = al.allocate<st_bf<ATTN_D, ROWS>, ACTIVE_TILES>(); //shared tile for debugging
    // st_bf<ATTN_D, ATTN_D> (&shared_debug_64)[ACTIVE_TILES]   = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES>(); //shared tile for debugging
    int total_block_idx = 0;

    if (warpid < ACTIVE_TILES + 1) {
        zero(s_s[warpid]);
    }
    //zero(s_s[warpid]);

    __syncthreads();

    
    int n_blocks = g.n / (ACTIVE_TILES * ROWS); // number of chunks we will loop over

    int i = 0;
    for (int block = 0; block < n_blocks; block++) {
        i += 1;
        // Load q, k, and v tiles in BF16 (global format: 16x64)
        rt_bf<ATTN_D, ONE> q, k;          // [16 x 64]
        rt_bf<ATTN_D, ONE> v;             // [16 x 64]
        rt_fl<ATTN_D, ONE> v_fl;          // [16 x 64] float version for arithmetic
        rt_fl<ATTN_D, ONE> o;             // [16 x 64] output

        rt_bf<1, ATTN_D> q_t, k_t;          // [16 x 64]
        rt_bf<1, ATTN_D> v_t;             // [16 x 64]
        //rt_fl<1, ATTN_D> v_fl;          // [16 x 64] float version for arithmetic
        rt_fl<1, ATTN_D> o_t;             // [16 x 64] output


        // Memory state (s_state) is 64x64 in float.
        rt_fl<ATTN_D, ATTN_D> s_state;     // [64 x 64] float
        rt_bf<ATTN_D, ATTN_D> s_state_bf;   // BF16 copy of s_state, [64 x 64]
        rt_fl<ATTN_D, ATTN_D> s_new;        // new memory state, [64 x 64] float
        rt_bf<ATTN_D, ATTN_D> s_new_bf;     // BF16 copy of s_new, [64 x 64]


        // Intermediate computation tiles (all [16x64]) in float
        rt_fl<ATTN_D, ONE> error;         // error = s_state*k^T - v, [16 x 64]
        rt_fl<ATTN_D, ONE> beta_error;    // [16 x 64] float
        rt_bf<ATTN_D, ONE> beta_error_bf; // BF16 version, [16 x 64]
        rt_fl<ATTN_D, ONE> P;             // [16 x 64] float

        // Outer product delta will be 64x64 in float and BF16.
        rt_fl<ATTN_D, ATTN_D> delta;       // [64 x 64] float
        rt_bf<ATTN_D, ATTN_D> delta_bf;    // [64 x 64] BF16

        zero(s_state);
        zero(s_new);
        zero(error);
        zero(beta_error);
        zero(P);//one(P); //one(P) gets rid of zeros
        zero(delta);
        // zero(shared_debug[warpid]);

        int cur_idx;

        if (warpid < ACTIVE_TILES) {
            // todo: set cur_idx to 0 and inspect first tile for q
            cur_idx = block * ACTIVE_TILES + warpid;
            load(qo_s[warpid], g.q, {batch, head, cur_idx, 0}); // ROWS x ATTN_D
            load(k_s[warpid], g.k, {batch, head, cur_idx, 0}); // ROWS x ATTN_D
            load(v_s[warpid], g.v, {batch, head, cur_idx, 0}); // ROWS x ATTN_D

        } else {
            // cur_idx = block * ACTIVE_TILES + warpid - ACTIVE_TILES;
            // load(v_s[warpid - ACTIVE_TILES], g.v, {batch, head, cur_idx, 0});
        }
        __syncthreads();

        // // --- Compute P = k * (s_state)^T ---
        if (warpid < ACTIVE_TILES) {
            
            load(q_t, qo_s[warpid]); // 1 x ATTN_D
            load(k_t, k_s[warpid]); // 1 x ATTN_D
            load(v_t, v_s[warpid]); // 1 x ATTN_D

            transpose_sep(q, q_t); // ATTN_D x 1
            transpose_sep(k, k_t); // ATTN_D x 1
            transpose_sep(v, v_t); // ATTN_D x 1

            zero(s_state); // ATTN_D x ATTN_D
            // check
            load(s_state, s_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]); // load current memory state

            copy(s_state_bf, s_state); // ATTN_D x ATTN_D

            auto & k_col = swap_layout_inplace(k);
            mma_AB(P, s_state_bf, k_col, P);
            // ATTN_D x 1 = // ATTN_D x ATTN_D * // ATTN_D x 1

            copy(error, P); // ATTN_D x 1
            copy(v_fl, v); // ATTN_D x 1
            sub(error, error, v_fl); // ATTN_D x 1

            copy(beta_error, error); // ATTN_D x 1
            mul(beta_error, beta_error, BETA); // ATTN_D x 1
            copy(beta_error_bf, beta_error); // ATTN_D x 1

            zero(delta); // ATTN_D x ATTN_D
            //auto & k_col = swap_layout_inplace(k); // 1 x ATTN_D
            mma_ABt(delta, beta_error_bf, k, delta); // we use mma_ABt so that it transposes k for us
            // ATTN_D x ATTN_D = // ATTN_D x 1 * 1 x ATTN_D

            copy(delta_bf, delta); // ATTN_D x ATTN_D
            copy(s_new, s_state); // ATTN_D x ATTN_D
            sub(s_new, s_new, delta); // ATTN_D x ATTN_D

            store(s_s[(total_block_idx + warpid + 1) % (ACTIVE_TILES + 1)], s_new); // ??
            copy(s_new_bf, s_new);
            auto & q_col = swap_layout_inplace(q);
            mma_AB(o, s_new_bf, q_col, o);
            // ATTN_D x 1 = // ATTN_D x ATTN_D * // ATTN_D x 1
            transpose_sep(o_t, o); // 1 x ATTN_D
            store(qo_s[warpid], o_t); // TODO fix layout for this

            }

            __syncthreads();


            cumsum_inplace<NUM_WORKERS>(s_s, total_block_idx);
            __syncthreads();

            if (warpid < ACTIVE_TILES) {
                rt_bf<ATTN_D, ATTN_D> s;
                load(q, qo_s[warpid]);
                load(s, s_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
                mma_ABt(o, q, s, o);
                // store(shared_debug_64[warpid], s);
                store(qo_s[warpid], o);
            }

        total_block_idx = (total_block_idx + ACTIVE_TILES) % (ACTIVE_TILES + 1);
        __syncthreads();

        if (warpid < ACTIVE_TILES) {
            //store(shared_debug[warpid], k);
            store(g.o, qo_s[warpid], {batch, head, cur_idx, 0});
            // store(g.o, v_s[warpid - ACTIVE_TILES], {batch, head, cur_idx, 0});
        }
        __syncthreads();
    }
}

fwd_globals fwd_init(
    bf16 *d_q, bf16 *d_k, bf16 *d_v,
    bf16 *d_o,
    long unsigned int ATTN_B, long unsigned int ATTN_H, long unsigned int ATTN_N
) {
    // global pointers
    std::cout << "init" << std::endl;
    using globals = fwd_globals;

    using q_tile     = globals::q_tile;
    using k_tile     = globals::k_tile;
    using v_tile     = globals::v_tile;
    using o_tile     = globals::o_tile;

    // global layouts
    using q_gl     = globals::q_gl;
    using k_gl     = globals::k_gl;
    using v_gl     = globals::v_gl;
    using o_gl     = globals::o_gl;

    q_gl     q_arg{d_q, ATTN_B, ATTN_H, ATTN_N, nullptr};
    k_gl     k_arg{d_k, ATTN_B, ATTN_H, ATTN_N, nullptr};
    v_gl     v_arg{d_v, ATTN_B, ATTN_H, ATTN_N, nullptr};
    o_gl     o_arg{d_o, ATTN_B, ATTN_H, ATTN_N, nullptr};

    globals g{
        q_arg, k_arg, v_arg, o_arg, ATTN_N
    };
    return g;
}

// ---------------------------------------------------------------------------------------------------
// ----------------------------------------- Backward kernel -----------------------------------------
// ---------------------------------------------------------------------------------------------------


struct bwd_globals {
    using q_tile = st_bf<ROWS, ATTN_D>;
    using k_tile = st_bf<ROWS, ATTN_D>;
    using v_tile = st_bf<ROWS, ATTN_D>;
    using do_tile = st_bf<ROWS, ATTN_D>;

    // global layouts
    using q_gl     = gl<bf16, -1, -1, -1, ATTN_D, q_tile>;
    using k_gl     = gl<bf16, -1, -1, -1, ATTN_D, k_tile>;
    using v_gl     = gl<bf16, -1, -1, -1, ATTN_D, v_tile>;
    using do_gl    = gl<bf16, -1, -1, -1, ATTN_D, do_tile>;

    // pointers
    q_gl q;
    k_gl k;
    v_gl v;
    do_gl d_o;

    q_gl dq;
    k_gl dk;
    v_gl dv;

    long unsigned int n;
};

// __global__ __launch_bounds__(NUM_THREADS, 1)
// void delta_attention_bwd(const __grid_constant__ bwd_globals g) {
    
//     const int batch = blockIdx.y;
//     const int head  = blockIdx.x;

//     int warpid = kittens::warpid(); 

//     extern __shared__ alignment_dummy __shm[]; 
//     shared_allocator al((int*)&__shm[0]);

//     st_bf<ROWS, ATTN_D> (&dodqqdk_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // do,dq for 1st loop, q,dk for 2nd loop
//     st_bf<ROWS, ATTN_D> (&k_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // k for 1st and 2nd
//     st_bf<ROWS, ATTN_D> (&v_s)[ACTIVE_TILES]   = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // v for 1st and 2nd
//     st_bf<ATTN_D, ATTN_D> (&hidden_s)[ACTIVE_TILES + 1]  = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES + 1>(); // accumulates hidden states (memory state S_t form forward pass
//     st_bf<ATTN_D, ATTN_D> (&dhidden_s)[ACTIVE_TILES + 1]  = al.allocate<st_bf<ATTN_D, ATTN_D>, ACTIVE_TILES + 1>(); // hidden state gradients (ds)
//     st_bf<ROWS, ATTN_D> (&dodv_s)[ACTIVE_TILES] = al.allocate<st_bf<ROWS, ATTN_D>, ACTIVE_TILES>(); // do,dv for 2nd

//     int total_block_idx = 0;

//     if (warpid < ACTIVE_TILES + 1) {
//         zero(hidden_s[warpid]); 
//         zero(dhidden_s[warpid]);
//     }
    
//     int n_blocks = g.n / (ACTIVE_TILES * ROWS);

//     // first loop: dq
//     for (int block = n_blocks - 1; block >= 0; block--) { // iterate backwards since S_t+1 impacts S_t
//         rt_bf<ROWS, ATTN_D> d_o, k, v;
//         rt_bf<ATTN_D, ROWS> vt;
//         rt_bf<ROWS, ROWS> local_attn_bf; //less precision long term storage
//         rt_fl<ROWS, ROWS> local_attn;
//         rt_fl<ATTN_D, ATTN_D> d_accum;
//         rt_fl<ROWS, ATTN_D> dq;

//         int cur_idx;

//         // load the data -> first half k, do, second half v 
//         if(warpid < ACTIVE_TILES) {
//             cur_idx = block * ACTIVE_TILES + warpid;
//             load(dodqqdk_s[warpid], g.d_o, {batch, head, cur_idx, 0});
//             load(k_s[warpid], g.k, {batch, head, cur_idx, 0});
//         }
//         else {
//             cur_idx = block * ACTIVE_TILES + warpid - ACTIVE_TILES;
//             load(v_s[warpid - ACTIVE_TILES], g.v, {batch, head, cur_idx, 0});
//         }
//         __syncthreads();

//         if (warpid < ACTIVE_TILES) {
//             load(d_o, dodqqdk_s[warpid]); // using dodqqdk_s as d_o
//             load(v, v_s[warpid]);

//             zero(local_attn);
//             mma_ABt(local_attn, d_o, v, local_attn); // local_attn <- d_o * v^T

//             // calculating the decay factor as a mask
//             rt_bf<ROWS, ROWS> decay;
//             zero(decay);

//             //calculaing the decay exponent as a lower triangular matrix
//             bf16* decay_ptr = reinterpret_cast<bf16*>(&decay);
//             #pragma unroll
//             for (int i = 0; i < ROWS; i++) {
//                 for (int j=0; j<ROWS; j++) {
//                     // decay.data[i*ROWS + j] *= __float2bfloat16(powf(BETA, i-j));
//                     decay_ptr[i * ROWS + j] = __float2bfloat16(powf(BETA, (float)(i - j)));
//                 }
//             }

//             // applying the decay factor to the local attention
//             bf16* local_attn_ptr = reinterpret_cast<bf16*>(&local_attn);
//             #pragma unroll
//             for (int i = 0; i < ROWS; i++) {
//                 for (int j=0; j <ROWS; j++) {
//                     local_attn_ptr[i * ROWS + j] *= decay_ptr[i * ROWS + j];
//                 }
//             }
            
//             //calculate local intra attention (will later accumulate over everything else)
//             zero(dq);
//             copy(local_attn_bf, local_attn);
//             load(k, k_s[warpid]);
//             auto &k_col = swap_layout_inplace(k);

//             mma_AB(dq, local_attn_bf, k_col, dq); // dq <- local_attn * k^T
            
//             //find dS
//             zero(d_accum);
//             mma_AtB(d_accum, v, k_col, d_accum); //or seperatly transpose v

//             //add gradient to the accumulated gradient
//             rt_fl<ATTN_D, ATTN_D> d_accum_loaded;
//             load(d_accum_loaded, d_accum[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);

//             bf16* d_accum_ptr = reinterpret_cast<bf16*>(&d_accum);
//             bf16* d_accum_loaded_ptr = reinterpret_cast<bf16*>(&d_accum_loaded);
//             #pragma unroll
//             for (int i=0; i<ATTN_D; i++) {
//                 for (int j=0; j<ATTN_D; j++) {
//                     d_accum_ptr[i*ATTN_D + j] += d_accum_ptr[i*ATTN_D + j] * BETA + d_accum_loaded_ptr[i*ATTN_D + j];
//                 }
//             }

//             store(dhidden_s[(total_block_idx + warpid + 1) % (ACTIVE_TILES + 1)], d_accum);
//         }

//         __syncthreads();
//         revcumsum_inplace<NUM_WORKERS>(dhidden_s, total_block_idx);
//         __syncthreads();

//         // doing actual dq calculation over the accumulated gradients
//         if (warpid < ACTIVE_TILES) {
//             rt_bf<ATTN_D, ATTN_D> gradient;
//             load(d_o, dodqqdk_s[warpid]);
//             load(gradient, dhidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
//             auto &gradient_col = swap_layout_inplace(gradient);
//             mma_AB(dq, d_o, gradient_col, dq);
//             store(dodqqdk_s[warpid], dq);
//         }

//         total_block_idx = (total_block_idx + ACTIVE_TILES) % (ACTIVE_TILES + 1);
//         __syncthreads();
        
//         if (warpid < ACTIVE_TILES) {
//             store(g.dq, dodqqdk_s[warpid], {batch, head, cur_idx, 0});
//         }
//         __syncthreads();
//     }

//     // second loop: dk, dv
//     total_block_idx = 0;

//     if (warpid < ACTIVE_TILES + 1) {
//         zero(hidden_s[warpid]);
//     }

//     for (int block = 0; block < n_blocks; block++) {
//         rt_bf<ROWS, ATTN_D> d_o, q, k, v;
//         rt_bf<ROWS, ATTN_D, col_l> q_col;
//         rt_bf<ATTN_D, ROWS> qt;
//         rt_bf<ROWS, ROWS> local_attn_bf;
//         rt_fl<ROWS, ROWS> local_attn;
//         rt_fl<ATTN_D, ATTN_D> dhidden_accum; // dS accumulated values
//         rt_fl<ROWS, ATTN_D> dk, dv;

//         int cur_idx;
//         if (warpid < ACTIVE_TILES) {
//             cur_idx = block * ACTIVE_TILES + warpid;
//             load(dodqqdk_s[warpid], g.d_o, {batch, head, cur_idx, 0});
//             load(k_s[warpid], g.k, {batch, head, cur_idx, 0});
//         }
//         else {
//             cur_idx = block * ACTIVE_TILES + warpid - ACTIVE_TILES;
//             load(v_s[warpid - ACTIVE_TILES], g.v, {batch, head, cur_idx, 0});
//             load(dodv_s[warpid - ACTIVE_TILES], g.d_o, {batch, head, cur_idx, 0});
//         }
//         __syncthreads();

//         if (warpid < ACTIVE_TILES) {
//             load(d_o, dodv_s[warpid]);
//             load(v, v_s[warpid]);

//             // dk calculaiton start
//             zero(local_attn);
//             mma_ABt(local_attn, v, d_o, local_attn); // local_attn <- v * d_o^T

//             rt_bf<ROWS, ROWS> decay;
//             zero(decay);

//             #pragma unroll
//             for (int i = 0; i < ROWS; i++) {
//                 for (int j=0; j<=i; j++) {
//                     decay.data[i*ROWS + j] *= powf(BETA, i-j)
//                 }
//             }

//             //applying mask to local delta calculated ealirer
//             #pragma unroll
//             for (int i = 0; i < ROWS; i++) {
//                 for (int j=0; j <ROWS; j++) {
//                     local_attn.data[i * ROWS + j] *= decay.data[i * ROWS + j];
//                 }
//             }

//             copy(local_attn_bf, local_attn);
//             load(q, dodqqdk_s[warpid]);
//             swap_layout(q_col, q);

//             zero(dk);
//             mma_AB(dk, local_attn_bf, q_col, dk); // dk <- local_attn * q^T 

//             // dv calculation start
//             load(k, k_s[warpid]);
//             zero(local_attn);
//             mma_ABt(local_attn, k, q, local_attn); // local_attn <- k * q^T

//             //applying decay
//             #pragma unroll
//             for (int i = 0; i < ROWS; i++) {
//                 for (int j=0; j< ROWS; j++) {
//                     local_attn.data[i * ROWS + j] *= decay.data[i * ROWS + j];
//                 }
//             }

//             copy(local_attn_bf, local_attn);

//             zero(dv);
//             auto &d_o_col = swap_layout_inplace(d_o);
//             mma_AB(dv, local_attn_bf, d_o_col, dv); // dv <- local_attn * d_o^T

//             // calculating S_t (hidden state)
//             transpose_sep(qt, q);
//             zero(dhidden_accum);
//             mma_AB(dhidden_accum, qt, d_o_col, dhidden_accum); // hidden_s <- q^T * d_o

//             //again applying decay factor
//             rt_fl<ATTN_D, ATTN_D> hidden_loaded;
//             load(hidden_loaded, dhidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);

//             #pragma unroll
//             for (int i=0; i<ATTN_D; i++) {
//                 for (int j=0; j<ATTN_D; j++) {
//                     dhidden_accum.data[i*ATTN_D + j] += hidden_loaded.data[i*ATTN_D + j] * BETA + dhidden_accum.data[i*ATTN_D + j];
//                 }
//             }
//             store(hidden_s[(total_block_idx + warpid + 1) % (ACTIVE_TILES + 1)], dhidden_accum);
//         }

//         __syncthreads();
//         cumsum_inplace<NUM_WORKERS>(hidden_s, total_block_idx);
//         __syncthreads();

//         // next part of dk calculation
//         if (warpid < ACTIVE_TILES) {
//             rt_bf<ATTN_D, ATTN_D> gradient;
//             load(v, v_s[warpid]);
//             load(gradient, hidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
//             auto &gradient_col = swap_layout_inplace(gradient);
//             mma_ABt(dk, v, gradient_col, dk);
//             store(dodqqdk_s[warpid], dk);
//         }

//         __syncthreads();

//         // next part of dv calculation
//         if (warpid < ACTIVE_TILES) {
//             rt_bf<ATTN_D, ATTN_D> gradient;
//             load(k, k_s[warpid]);
//             load(gradient, hidden_s[(total_block_idx + warpid) % (ACTIVE_TILES + 1)]);
//             auto &gradient_col = swap_layout_inplace(gradient);
//             mma_AB(dv, k, gradient, dv);
//             store(dodv_s[warpid], dv);
//         }

//         total_block_idx = (total_block_idx + ACTIVE_TILES) % (ACTIVE_TILES + 1);
//         __syncthreads();

//         if (warpid < ACTIVE_TILES) {
//             store(g.dk, dodqqdk_s[warpid], {batch, head, cur_idx, 0});
//             store(g.dv, dodv_s[warpid], {batch, head, cur_idx, 0});
//         }
//         __syncthreads();
//     }

// }

bwd_globals bwd_init(
    bf16 *d_q, bf16 *d_k, bf16 *d_v, bf16 *d_do,
    bf16 *d_dq, bf16 *d_dk, bf16 *d_dv,
    long unsigned int ATTN_B, long unsigned int ATTN_H, long unsigned int ATTN_N
) {
    // global pointers

    using globals = bwd_globals;

    using q_tile     = globals::q_tile;
    using k_tile     = globals::k_tile;
    using v_tile     = globals::v_tile;
    using do_tile     = globals::do_tile;

    // global layouts
    globals::q_gl  q_arg{d_q, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::k_gl  k_arg{d_k, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::v_gl  v_arg{d_v, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::do_gl do_arg{d_do, ATTN_B, ATTN_H, ATTN_N, nullptr};

    globals::q_gl dq_arg{d_dq, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::k_gl dk_arg{d_dk, ATTN_B, ATTN_H, ATTN_N, nullptr};
    globals::v_gl dv_arg{d_dv, ATTN_B, ATTN_H, ATTN_N, nullptr};

    globals g{
        q_arg, k_arg, v_arg, do_arg, dq_arg, dk_arg, dv_arg, ATTN_N
    };
    return g;
}


#ifdef TK_COMPILE_LIN_ATTN
#include "pyutils/torch_helpers.cuh"
#include <iostream>
void dispatch_fwd( 
    bf16 *d_q, bf16 *d_k, bf16 *d_v, bf16 *d_o,
    int ATTN_B, int ATTN_H, int ATTN_N
){
    fwd_globals g = fwd_init(
        d_q, d_k, d_v,
        d_o,
        ATTN_B, ATTN_H, ATTN_N
    );

    // launch
    unsigned long mem_size = 500000; // 4090
    hipDeviceSynchronize();
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        delta_attention_fwd),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    dim3 grid(ATTN_H, ATTN_B);
    delta_attention_fwd<<<grid,NUM_THREADS,mem_size>>>(g);
    CHECK_CUDA_ERROR(hipGetLastError());
    hipDeviceSynchronize();
}

torch::Tensor delta_attn_forward(
    const torch::Tensor q, 
    const torch::Tensor k,
    const torch::Tensor v
) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);

    int B = q.size(0);
    int H = q.size(1);
    int DV = v.size(3);
    int N  = q.size(2);
    int FD = k.size(3);

    // checks
    TORCH_CHECK(k.size(0) == B, "k batch?");
    TORCH_CHECK(k.size(1) == H, "k heads?");
    TORCH_CHECK(k.size(2) == N, "k length?");

    TORCH_CHECK(v.size(0) == B, "v batch?");
    TORCH_CHECK(v.size(1) == H, "v heads?");
    TORCH_CHECK(v.size(2) == N, "v length?");

    // allocate output
    torch::Tensor out = torch::empty({B, H, N, DV}, v.options());

    // convert to bf16
    c10::BFloat16 *q_bf16 = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_bf16 = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_bf16 = v.data_ptr<c10::BFloat16>();
    
    bf16 *d_q = reinterpret_cast<bf16*>(q_bf16);
    bf16 *d_k = reinterpret_cast<bf16*>(k_bf16);
    bf16 *d_v = reinterpret_cast<bf16*>(v_bf16);
    bf16 *d_o = reinterpret_cast<bf16*>(out.data_ptr<c10::BFloat16>());

    dispatch_fwd(
        d_q, d_k, d_v, d_o,
        B, H, N
    );

    CHECK_CUDA_ERROR(hipGetLastError());
    return out;
    hipDeviceSynchronize();
}

void dispatch_bwd(
    bf16 *d_q, bf16 *d_k, bf16 *d_v, bf16 *d_do,
    bf16 *d_dq, bf16 *d_dk, bf16 *d_dv,
    int ATTN_B, int ATTN_H, int ATTN_N
){
    bwd_globals g = bwd_init(
        d_q, d_k, d_v, d_do,
        d_dq, d_dk, d_dv,
        ATTN_B, ATTN_H, ATTN_N
    );

    // launch
    unsigned long mem_size = 100000; // 4090
    hipDeviceSynchronize();
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        delta_attention_bwd),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    dim3 grid(ATTN_H, ATTN_B);
    delta_attention_bwd<<grid,NUM_THREADS,mem_size>>>(g);
    CHECK_CUDA_ERROR(hipGetLastError());
    hipDeviceSynchronize();
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> delta_attn_backward(
    const torch::Tensor q, 
    const torch::Tensor k,
    const torch::Tensor v,
    const torch::Tensor _do
) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(_do);

    int B = q.size(0);
    int H = q.size(1);
    int DV = v.size(3);
    int N  = q.size(2);
    int FD = k.size(3);

    // checks
    TORCH_CHECK(k.size(0) == B, "k batch?");
    TORCH_CHECK(k.size(1) == H, "k heads?");
    TORCH_CHECK(k.size(2) == N, "k length?");

    TORCH_CHECK(v.size(0) == B, "v batch?");
    TORCH_CHECK(v.size(1) == H, "v heads?");
    TORCH_CHECK(v.size(2) == N, "v length?");

    // allocate output
    torch::Tensor out_dq = torch::empty({B, H, N, FD}, q.options());
    torch::Tensor out_dk = torch::empty({B, H, N, FD}, k.options());
    torch::Tensor out_dv = torch::empty({B, H, N, DV}, v.options());

    // convert to bf16
    c10::BFloat16 *q_bf16 = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_bf16 = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_bf16 = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *do_bf16 = _do.data_ptr<c10::BFloat16>();
    
    bf16 *d_q = reinterpret_cast<bf16*>(q_bf16);
    bf16 *d_k = reinterpret_cast<bf16*>(k_bf16);
    bf16 *d_v = reinterpret_cast<bf16*>(v_bf16);
    bf16 *d_do = reinterpret_cast<bf16*>(do_bf16);
    bf16 *d_dq = reinterpret_cast<bf16*>(out_dq.data_ptr<c10::BFloat16>());
    bf16 *d_dk = reinterpret_cast<bf16*>(out_dk.data_ptr<c10::BFloat16>());
    bf16 *d_dv = reinterpret_cast<bf16*>(out_dv.data_ptr<c10::BFloat16>());

    dispatch_bwd(
        d_q, d_k, d_v, d_do,
        d_dq, d_dk, d_dv,
        B, H, N
    );

    CHECK_CUDA_ERROR(hipGetLastError());
    return std::make_tuple(out_dq, out_dk, out_dv);
    hipDeviceSynchronize();
}

// #else
// #ifdef FWD_HARNESS
// #include "4090_harness_fwd.impl"
// #else
// #include "4090_harness_bwd.impl"
// #endif
#endif
#include "fwd_harness.impl"