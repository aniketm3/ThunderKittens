#include "hip/hip_runtime.h"
#include "kittens.cuh"

using namespace kittens;

constexpr int NUM_WORKERS = 4; // This kernel uses 4 worker warps per block, and 2 blocks per SM.
template<int D> constexpr size_t ROWS = 16*(128/D); // height of each worker tile (rows)
template<int D, typename T=bf16, typename L=row_l> using qkvo_tile = rt<T, ROWS<D>, D, L>;
template<int D, typename T=float> using attn_tile = rt<T, ROWS<D>, ROWS<D>>;
template<int D> using shared_tile = st_bf<ROWS<D>, D>;
template<int D> using global_layout = gl<bf16, -1, -1, -1, D>; // B, N, H, specified at runtime, D known at compile time for this kernel
template<int D> struct globals { global_layout<D> Qg, Kg, Vg, Og; };

template<int D> __launch_bounds__(NUM_WORKERS*WARP_THREADS, 1)
__global__ void attend_ker(const __grid_constant__ globals<D> g) {
    using load_group = kittens::group<2>; // pairs of workers collaboratively load k, v tiles
    int loadid = load_group::groupid(), workerid = kittens::warpid(); // which worker am I?
    constexpr int LOAD_BLOCKS = NUM_WORKERS / load_group::GROUP_WARPS;
    const int batch = blockIdx.z, head = blockIdx.y, q_seq = blockIdx.x * NUM_WORKERS + workerid;

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    // K and V live in shared memory. Here, we instantiate three tiles for a 3-stage pipeline.
    shared_tile<D> (&k_smem)[LOAD_BLOCKS][3] = al.allocate<shared_tile<D>, LOAD_BLOCKS, 3>();
    shared_tile<D> (&v_smem)[LOAD_BLOCKS][3] = al.allocate<shared_tile<D>, LOAD_BLOCKS, 3>();
    // We also reuse this memory to improve coalescing of DRAM reads and writes.
    shared_tile<D> (&qo_smem)[NUM_WORKERS] = reinterpret_cast<shared_tile<D>(&)[NUM_WORKERS]>(k_smem);
    // Initialize all of the register tiles.
    qkvo_tile<D, bf16> q_reg, k_reg; // Q and K are both row layout, as we use mma_ABt.
    qkvo_tile<D, bf16, col_l> v_reg; // V is column layout, as we use mma_AB.
    qkvo_tile<D, float> o_reg; // Output tile.
    attn_tile<D, float> att_block; // attention tile, in float. (We want to use float wherever possible.)
    attn_tile<D, bf16> att_block_mma; // bf16 attention tile for the second mma_AB. We cast right before that op.
    typename attn_tile<D, float>::col_vec max_vec_last, max_vec, norm_vec; // these are column vectors for the in-place softmax.
    // each warp loads its own Q tile of 16x64
    if (q_seq*ROWS<D> < g.Qg.depth) {
        load<shared_tile<D>, global_layout<D>, 1>(qo_smem[workerid], g.Qg, {batch, q_seq, head, 0});  // going through shared memory improves coalescing of dram reads.
        __syncwarp();
        load(q_reg, qo_smem[workerid]);
    }
    __syncthreads();
    // temperature adjustment. Pre-multiplying by lg2(e), too, so we can use exp2 later.
    if constexpr(D == 64) mul(q_reg, q_reg, __float2bfloat16(0.125f * 1.44269504089));
    else if constexpr(D == 128) mul(q_reg, q_reg, __float2bfloat16(0.08838834764f * 1.44269504089));
    // initialize flash attention L, M, and O registers.
    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_reg);
    // launch the load of the first k, v tiles
    int kv_blocks = g.Kg.depth / (LOAD_BLOCKS*ROWS<D>), tic = 0;
    load_group::load_async<shared_tile<D>, global_layout<D>, 1>(k_smem[loadid][0], g.Kg, {batch, loadid, head, 0});
    load_group::load_async<shared_tile<D>, global_layout<D>, 1>(v_smem[loadid][0], g.Vg, {batch, loadid, head, 0});
    // iterate over k, v for these q's that have been loaded
    for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic=(tic+1)%3) {
        int next_load_idx = (kv_idx+1)*LOAD_BLOCKS + loadid;
        if(next_load_idx*ROWS<D> < g.Kg.depth) {
            int next_tic = (tic+1)%3;
            load_group::load_async<shared_tile<D>, global_layout<D>, 1>(k_smem[loadid][next_tic], g.Kg, {batch, next_load_idx, head, 0});
            load_group::load_async<shared_tile<D>, global_layout<D>, 1>(v_smem[loadid][next_tic], g.Vg, {batch, next_load_idx, head, 0});
            load_async_wait<1>(); // next k, v can stay in flight.
        }
        else load_async_wait(); // all must arrive
        __syncthreads(); // Everyone's memory must be ready for the next stage.
        // now each warp goes through all of the subtiles, loads them, and then does the flash attention internal alg.
        #pragma unroll LOAD_BLOCKS
        for(int subtile = 0; subtile < LOAD_BLOCKS && (kv_idx*LOAD_BLOCKS + subtile) < g.Kg.depth/ROWS<D>; subtile++) {
            load(k_reg, k_smem[subtile][tic]); // load k from shared into registers
            zero(att_block); // zero 16x16 attention tile
            mma_ABt(att_block, q_reg, k_reg, att_block); // Q@K.T
            copy(max_vec_last,  max_vec);
            row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
            sub_row(att_block, att_block, max_vec); // subtract max from attention -- now all <=0
            exp2(att_block, att_block); // exponentiate the block in-place.
            sub(max_vec_last, max_vec_last, max_vec); // subtract new max from old max to find the new normalization.
            exp2(max_vec_last, max_vec_last); // exponentiate this vector -- this is what we need to normalize by.
            mul(norm_vec, norm_vec, max_vec_last); // and the norm vec is now normalized.
            row_sum(norm_vec, att_block, norm_vec); // accumulate the new attention block onto the now-rescaled norm_vec
            copy(att_block_mma, att_block); // convert to bf16 for mma_AB
            load(v_reg, v_smem[subtile][tic]); // load v from shared into registers.
            mul_row(o_reg, o_reg, max_vec_last); // normalize o_reg in advance of mma_AB'ing onto it
            mma_AB(o_reg, att_block_mma, v_reg, o_reg); // mfma onto o_reg with the local attention@V matmul.
        }
    }
    div_row(o_reg, o_reg, norm_vec);
    __syncthreads();
    if (q_seq*ROWS<D> < g.Og.depth) { // write out o.
        store(qo_smem[workerid], o_reg); // going through shared memory improves coalescing of dram writes.
        __syncwarp();
        store<shared_tile<D>, global_layout<D>, 1>(g.Og, qo_smem[workerid], {batch, q_seq, head, 0});
    }
}

#include "4090_harness.impl"