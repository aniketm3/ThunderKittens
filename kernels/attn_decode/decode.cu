#include "hip/hip_runtime.h"
#define TORCH_COMPILE
#include "kittens.cuh"

using namespace kittens;

constexpr int NUM_WORKERS = 4; // This kernel uses 4 worker warps per block, and 2 blocks per SM.
template<int D> constexpr size_t ROWS = 16*(128/D); // height of each worker tile (rows)
template<int D, typename T=bf16, typename L=row_l> using qkvo_tile = rt<T, ROWS<D>, D, L>;
template<int D, typename T=float> using attn_tile = rt<T, ROWS<D>, ROWS<D>>;
template<int D> using shared_tile = st_bf<ROWS<D>, D>;
template<int D> using global_layout = gl<bf16, -1, -1, -1, D>; // B, H, g.Qg.rows specified at runtime, D=64 known at compile time for this kernel
template<int D> struct globals { global_layout<D> Qg, KCacheg, VCacheg, Og, KNewg, VNewg; };

template<int D> __launch_bounds__(NUM_WORKERS*WARP_THREADS, 1)
__global__ void attend_ker(
    const __grid_constant__ globals<D> g,   // Q, KCache, VCache, O, KNew, VNew
    int k_seqlen,                           // KCache sequence length
    int k_new_seqlen,                       // KNew sequence length
    bool causal                             // causal attention flag
) {
    auto ZERO = kittens::base_types::constants<bf16>::zero();
    using load_group = kittens::group<2>; // pairs of workers collaboratively load k, v tiles
    int loadid = load_group::groupid(), workerid = kittens::warpid(); // which worker am I?
    constexpr int LOAD_BLOCKS = NUM_WORKERS / load_group::GROUP_WARPS;
    const int batch = blockIdx.z, head  = blockIdx.y, q_seq = blockIdx.x * NUM_WORKERS + workerid;
    const int q_seq_next = (blockIdx.x + 1) * NUM_WORKERS;

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    // K and V live in shared memory. Here, we instantiate three tiles for a 3-stage pipeline.
    shared_tile<D> (&k_smem)[LOAD_BLOCKS][3] = al.allocate<shared_tile<D>, LOAD_BLOCKS, 3>();
    shared_tile<D> (&v_smem)[LOAD_BLOCKS][3] = al.allocate<shared_tile<D>, LOAD_BLOCKS, 3>();
    // We also reuse this memory to improve coalescing of DRAM reads and writes.
    shared_tile<D> (&qo_smem)[NUM_WORKERS] = reinterpret_cast<shared_tile<D>(&)[NUM_WORKERS]>(k_smem);
    // Initialize all of the register tiles.
    qkvo_tile<D, bf16> q_reg, k_reg; // Q and K are both row layout, as we use mma_ABt.
    qkvo_tile<D, bf16, col_l> v_reg; // V is column layout, as we use mma_AB.
    qkvo_tile<D, float> o_reg; // Output tile.
    attn_tile<D, float> att_block; // attention tile, in float. (We want to use float wherever possible.)
    attn_tile<D, bf16> att_block_mma; // bf16 attention tile for the second mma_AB. We cast right before that op.
    typename attn_tile<D, float>::col_vec max_vec_last, max_vec, norm_vec; // these are column vectors for the in-place softmax.
    // each warp loads its own Q tile of 16x64
    if (q_seq*ROWS<D> < g.Qg.rows) {
        load<shared_tile<D>, global_layout<D>, 2>(qo_smem[workerid], g.Qg, {batch, head, q_seq, 0}, ROWS<D>, ZERO);  // going through shared memory improves coalescing of dram reads.
        __syncwarp();
        load(q_reg, qo_smem[workerid]);
    }
    __syncthreads();
    // temperature adjustment. Pre-multiplying by lg2(e), too, so we can use exp2 later.
    if constexpr(D == 64) mul(q_reg, q_reg, __float2bfloat16(0.125f * 1.44269504089));
    else if constexpr(D == 128) mul(q_reg, q_reg, __float2bfloat16(0.08838834764f * 1.44269504089));
    // initialize flash attention L, M, and O registers.
    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_reg);
    // launch the load of the first k, v tiles
    // total number of blocks we want to load
    int kv_blocks = (k_seqlen + (LOAD_BLOCKS*ROWS<D>) - 1) / (LOAD_BLOCKS*ROWS<D>);
    int kv_blocks_total = (k_seqlen + k_new_seqlen + (LOAD_BLOCKS*ROWS<D>) - 1) / (LOAD_BLOCKS*ROWS<D>);
    int tic = 0;
    load_group::load_async<shared_tile<D>, global_layout<D>, 2>(k_smem[loadid][0], g.KCacheg, {batch, head, loadid, 0}, ROWS<D>, ZERO);
    load_group::load_async<shared_tile<D>, global_layout<D>, 2>(v_smem[loadid][0], g.VCacheg, {batch, head, loadid, 0}, ROWS<D>, ZERO);
    // iterate over k, v for these q's that have been loaded
    for(auto kv_idx = 0; kv_idx < kv_blocks_total; kv_idx++, tic=(tic+1)%3) {
        int cur_load_idx = kv_idx*LOAD_BLOCKS;

        int next_load_idx = (kv_idx+1)*LOAD_BLOCKS + loadid;
        bool load_next = true;
        if (k_new_seqlen == 0) {
            // skip if we're out of K's or we're past the causal mask
            load_next = next_load_idx*ROWS<D> < k_seqlen && (!causal || next_load_idx <= q_seq_next);
        } else {
            // skip if we're out of KNew's or we're past the causal mask for KNew
            load_next = next_load_idx*ROWS<D> < k_seqlen + k_new_seqlen && (!causal || (kv_idx + 1 - kv_blocks) * LOAD_BLOCKS + loadid <= q_seq_next);
        }
        if(load_next && next_load_idx*ROWS<D> < k_seqlen) {
            // every two workers are working together to load the next tiles, then broadcast to all workers
            // we need to load the next times for all workers, and then skip selectively in the individual worker
            int next_tic = (tic+1)%3;
            load_group::load_async<shared_tile<D>, global_layout<D>, 2>(k_smem[loadid][next_tic], g.KCacheg, {batch, head, next_load_idx, 0}, ROWS<D>, ZERO);
            load_group::load_async<shared_tile<D>, global_layout<D>, 2>(v_smem[loadid][next_tic], g.VCacheg, {batch, head, next_load_idx, 0}, ROWS<D>, ZERO);
            load_async_wait<2>(); // next k, v can stay in flight.
        }
        else if (load_next && next_load_idx*ROWS<D> < k_seqlen + k_new_seqlen) {
            // load the next tiles from KNew and VNew
            int next_tic = (tic+1)%3;
            int kv_new_idx = (kv_idx - kv_blocks + 1) * LOAD_BLOCKS + loadid;
            load_group::load_async<shared_tile<D>, global_layout<D>, 2>(k_smem[loadid][next_tic], g.KNewg, {batch, head, kv_new_idx, 0}, ROWS<D>, ZERO);
            load_group::load_async<shared_tile<D>, global_layout<D>, 2>(v_smem[loadid][next_tic], g.VNewg, {batch, head, kv_new_idx, 0}, ROWS<D>, ZERO);
            load_async_wait<2>(); // next k, v can stay in flight.
        }
        else load_async_wait(); // all must arrive
        __syncthreads(); // Everyone's memory must be ready for the next stage.
        // now each warp goes through all of the subtiles, loads them, and then does the flash attention internal alg.
        #pragma unroll LOAD_BLOCKS
        for(int subtile = 0;
            subtile < LOAD_BLOCKS &&
            kv_idx * LOAD_BLOCKS * ROWS<D> + subtile * ROWS<D> < k_seqlen + k_new_seqlen;
            subtile++) {
            if (
                causal && (
                    (
                        k_new_seqlen == 0 &&
                        kv_idx * LOAD_BLOCKS * ROWS<D> + subtile * ROWS<D> > q_seq * ROWS<D> // we've passed the diagonal for this subtile and not using KNew
                    ) ||
                    (
                        k_new_seqlen > 0 &&
                        kv_idx >= kv_blocks &&
                        (kv_idx - kv_blocks) * LOAD_BLOCKS * ROWS<D> + subtile * ROWS<D> > q_seq * ROWS<D> // we've passed the diagonal for this subtile and using KNew
                    )
                )
            ){
                break;
            }

            load(k_reg, k_smem[subtile][tic]); // load k from shared into registers
            zero(att_block); // zero 16x16 attention tile
            mma_ABt(att_block, q_reg, k_reg, att_block); // Q@K.T
            if (
                causal && (
                    (
                        k_new_seqlen == 0 && 
                        kv_idx * LOAD_BLOCKS * ROWS<D> + subtile * ROWS<D> == q_seq * ROWS<D>
                    ) // we are not using KNew, and Q_tile == K_tile on the diagonal
                    || (
                        k_new_seqlen > 0 &&
                        kv_idx >= kv_blocks &&
                        ((kv_idx - kv_blocks) * LOAD_BLOCKS * ROWS<D> + subtile * ROWS<D> == q_seq * ROWS<D>)
                    ) // we are using KNew, and Q_tile == K_new_tile, on the diagonal
                )
            ) {
                // mask out the upper triangle
                make_causal(att_block, att_block, kittens::base_types::constants<float>::neg_infty());
            }
            copy(max_vec_last,  max_vec);
            row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
            sub_row(att_block, att_block, max_vec); // subtract max from attention -- now all <=0
            exp2(att_block, att_block); // exponentiate the block in-place.
            sub(max_vec_last, max_vec_last, max_vec); // subtract new max from old max to find the new normalization.
            exp2(max_vec_last, max_vec_last); // exponentiate this vector -- this is what we need to normalize by.
            mul(norm_vec, norm_vec, max_vec_last); // and the norm vec is now normalized.
            row_sum(norm_vec, att_block, norm_vec); // accumulate the new attention block onto the now-rescaled norm_vec
            copy(att_block_mma, att_block); // convert to bf16 for mma_AB
            load(v_reg, v_smem[subtile][tic]); // load v from shared into registers.
            mul_row(o_reg, o_reg, max_vec_last); // normalize o_reg in advance of mma_AB'ing onto it
            mma_AB(o_reg, att_block_mma, v_reg, o_reg); // mfma onto o_reg with the local attention@V matmul.
        }
    }
    div_row(o_reg, o_reg, norm_vec);
    __syncthreads();
    if (q_seq*ROWS<D> < g.Qg.rows) { // write out o.
        store(qo_smem[workerid], o_reg); // going through shared memory improves coalescing of dram writes.
        __syncwarp();
        store<shared_tile<D>, global_layout<D>, 2>(g.Og, qo_smem[workerid], {batch, head, q_seq, 0});

        if (k_new_seqlen > 0) {
            __syncwarp();
            int kv_blocks_orig = (k_seqlen + ROWS<D> - 1) / ROWS<D>;
            // in-place update KCache with KNew
            load<shared_tile<D>, global_layout<D>, 2>(qo_smem[workerid], g.KNewg, {batch, head, q_seq, 0}, ROWS<D>, ZERO);  // going through shared memory improves coalescing of dram reads.
            __syncwarp();
            store<shared_tile<D>, global_layout<D>, 2>(g.KCacheg, qo_smem[workerid], {batch, head, kv_blocks_orig + q_seq, 0});

            __syncwarp();
            // in-place update VCache with VNew
            load<shared_tile<D>, global_layout<D>, 2>(qo_smem[workerid], g.VNewg, {batch, head, q_seq, 0}, ROWS<D>, ZERO);  // going through shared memory improves coalescing of dram reads.
            __syncwarp();
            store<shared_tile<D>, global_layout<D>, 2>(g.VCacheg, qo_smem[workerid], {batch, head, kv_blocks_orig + q_seq, 0});
        }
    }
}

#ifdef TORCH_COMPILE

#include "common/pyutils/torch_helpers.cuh"
#include <ATen/cuda/HIPContext.h>
#include <iostream>

/**
 * @brief Decode attention forward pass with a KV cache and optional in-place update to the KV cache.
 * 
 * @param q The new query. (batch, num_heads, seqlen_q, head_dim)
 * @param k_cache The existing key cache. (batch, num_heads, seqlen_k_max, head_dim)
 * @param v_cache The existing value cache. (batch, num_heads, seqlen_v, head_dim)
 * @param k_new_ The new keys to update k_cache in-place. (batch, num_heads, seqlen_q, head_dim)
 * @param v_new_ The new values to update v_cache in-place. (batch, num_heads, seqlen_q, head_dim)
 * @param causal Whether to use causal attention. If k_new_ and v_new_ are provided, causal mask is only applied against the new queries and keys. If they are not provided, k_seqlen must match seqlen_q, and the causal mask is applied against all the queries and keys.
 * @param k_seqlen The sequence length of the key cache.
 * @return The output of the attention operation. (batch, num_heads, seqlen_q, head_dim)
 */
torch::Tensor
attention_decode_forward(
    torch::Tensor q,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    c10::optional<torch::Tensor> k_new_,
    c10::optional<torch::Tensor> v_new_,
    bool causal,
    int k_seqlen
)
{
    CHECK_INPUT(q);
    CHECK_INPUT(k_cache);
    CHECK_INPUT(v_cache);

    auto batch     = q.size(0);
    auto q_seq_len = q.size(2); 
    auto k_max_len = k_cache.size(2); 
    auto head_dim  = q.size(3); 
    auto qo_heads  = q.size(1);
    auto kv_heads  = k_cache.size(1);

    TORCH_CHECK(k_seqlen % 32 == 0, "K sequence length must be divisible by 32");

    // check to see that these dimensions match for all inputs
    TORCH_CHECK(q.size(0) == batch, "Q batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(k_cache.size(0) == batch, "K cache batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(v_cache.size(0) == batch, "V cache batch dimension - idx 0 - must match for all inputs");

    TORCH_CHECK(q_seq_len % 32 == 0, "Q sequence length must be divisible by 32");
    TORCH_CHECK(k_max_len % 32 == 0, "K cache sequence length must be divisible by 32");

    TORCH_CHECK(v_cache.size(2) == k_max_len, "V cache sequence length dimension - idx 2 - must match for all inputs");

    TORCH_CHECK(q.size(3) == head_dim, "Q head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(k_cache.size(3) == head_dim, "K cache head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(v_cache.size(3) == head_dim, "V cache head dimension - idx 3 - must match for all non-vector inputs");

    TORCH_CHECK(qo_heads >= kv_heads, "QO heads must be greater than or equal to KV heads");
    TORCH_CHECK(qo_heads % kv_heads == 0, "QO heads must be divisible by KV heads");
    TORCH_CHECK(q.size(1) == qo_heads, "QO head dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(k_cache.size(1) == kv_heads, "KV head dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(v_cache.size(1) == kv_heads, "KV head dimension - idx 1 - must match for all inputs");

    torch::Tensor k_new, v_new;
    auto k_new_seqlen = k_new_.has_value() ? k_new_.value().size(2) : 0;
    if (k_new_.has_value()) {
        assert(v_new_.has_value());
        k_new = k_new_.value();
        v_new = v_new_.value();
        CHECK_INPUT(k_new);
        CHECK_INPUT(v_new);
        TORCH_CHECK(k_new.size(0) == batch, "K new batch dimension - idx 0 - must match for all inputs");
        TORCH_CHECK(v_new.size(0) == batch, "V new batch dimension - idx 0 - must match for all inputs");
        TORCH_CHECK(k_new.size(1) == kv_heads, "K new heads - idx 1 - must match for all inputs");
        TORCH_CHECK(v_new.size(1) == kv_heads, "V new heads - idx 1 - must match for all inputs");
        TORCH_CHECK(k_new.size(2) == q_seq_len, "K new sequence length - idx 2 - must match for all inputs");
        TORCH_CHECK(v_new.size(2) == q_seq_len, "V new sequence length - idx 2 - must match for all inputs");
        TORCH_CHECK(k_new.size(3) == head_dim, "K new head dimension - idx 3 - must match for all inputs");
        TORCH_CHECK(v_new.size(3) == head_dim, "V new head dimension - idx 3 - must match for all inputs");
    }
    
    auto hr = qo_heads / kv_heads;

    c10::BFloat16* q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16* k_cache_ptr = k_cache.data_ptr<c10::BFloat16>();
    c10::BFloat16* v_cache_ptr = v_cache.data_ptr<c10::BFloat16>();
    c10::BFloat16* k_new_ptr = k_new_.has_value() ? k_new.data_ptr<c10::BFloat16>() : nullptr;
    c10::BFloat16* v_new_ptr = v_new_.has_value() ? v_new.data_ptr<c10::BFloat16>() : nullptr;

    bf16*  d_q = reinterpret_cast<bf16*>(q_ptr);
    bf16*  d_k_cache = reinterpret_cast<bf16*>(k_cache_ptr);
    bf16*  d_v_cache = reinterpret_cast<bf16*>(v_cache_ptr);
    bf16*  d_k_new = k_new_ptr ? reinterpret_cast<bf16*>(k_new_ptr) : nullptr;
    bf16*  d_v_new = v_new_ptr ? reinterpret_cast<bf16*>(v_new_ptr) : nullptr;
    
    // for the returned outputs
    torch::Tensor o     = torch::empty({static_cast<const uint>(batch), 
                                        static_cast<const uint>(qo_heads), 
                                        static_cast<const uint>(q_seq_len), 
                                        static_cast<const uint>(head_dim)}, q.options());
    
    bf16*  o_ptr = reinterpret_cast<bf16*>(o.data_ptr<c10::BFloat16>());
    bf16*  d_o   = reinterpret_cast<bf16*>(o_ptr);

    hipDeviceSynchronize();

    unsigned long mem_size = (kittens::MAX_SHARED_MEMORY-1000) / 2; // have the flag tell us

    if (head_dim == 64) {
        global_layout<64> qg(d_q, batch, qo_heads, q_seq_len, nullptr);
        global_layout<64> kg(d_k_cache, batch, kv_heads, k_max_len, nullptr);
        global_layout<64> vg(d_v_cache, batch, kv_heads, k_max_len, nullptr);
        global_layout<64> og(d_o, batch, qo_heads, q_seq_len, nullptr);
        global_layout<64> kg_new(d_k_new, batch, kv_heads, q_seq_len, nullptr);
        global_layout<64> vg_new(d_v_new, batch, kv_heads, q_seq_len, nullptr);
        globals<64> g(qg, kg, vg, og, kg_new, vg_new);

        hipFuncSetAttribute(reinterpret_cast<const void*>(
            attend_ker<64>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            mem_size
        );

        dim3 grid((q_seq_len + qkvo_tile<64>::rows*NUM_WORKERS - 1) / (qkvo_tile<64>::rows*NUM_WORKERS), qo_heads, batch);
        attend_ker<64><<<grid, (32*NUM_WORKERS), mem_size>>>(
            g,
            k_seqlen,
            k_new_seqlen,
            causal
        );
    }
    else if (head_dim == 128) {
        global_layout<128> qg(d_q, batch, qo_heads, q_seq_len, nullptr);
        global_layout<128> kg(d_k_cache, batch, kv_heads, k_max_len, nullptr);
        global_layout<128> vg(d_v_cache, batch, kv_heads, k_max_len, nullptr);
        global_layout<128> og(d_o, batch, qo_heads, q_seq_len, nullptr);
        global_layout<128> kg_new(d_k_new, batch, kv_heads, q_seq_len, nullptr);
        global_layout<128> vg_new(d_v_new, batch, kv_heads, q_seq_len, nullptr);
        globals<128> g(qg, kg, vg, og, kg_new, vg_new);

        hipFuncSetAttribute(reinterpret_cast<const void*>(
            attend_ker<128>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            mem_size
        );

        dim3 grid((q_seq_len + qkvo_tile<128>::rows*NUM_WORKERS - 1) / (qkvo_tile<128>::rows*NUM_WORKERS), qo_heads, batch);
        attend_ker<128><<<grid, (32*NUM_WORKERS), mem_size>>>(
            g,
            k_seqlen,
            k_new_seqlen,
            causal
        );
    }
    else {
        TORCH_CHECK(false, "head_dim must be 64 or 128");
    }

    CHECK_CUDA_ERROR(hipGetLastError());

    return o;
    hipDeviceSynchronize();
}

#endif